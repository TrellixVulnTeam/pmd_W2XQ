#include "hip/hip_runtime.h"
#include "pairwise_l1.h"
#include "math.h"
#include "assert.h"

// Matrix multiplication kernel called by MatMul()
__global__ void PairwiseL1Kernel(float *A, float *B, float *C, int n, int m, int d) {
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= n || col >= m) return;
    for (int e = 0; e < d; ++e)
        Cvalue += fabs(A[row * d + e] - B[col * d + e]);
    C[row * m + col] = Cvalue;
}

float *d_A = 0;
float *d_B = 0;
float *d_C = 0;

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void PairwiseL1(float *A, float *B, float *C, int n, int m, int d) {
    // Load A and B to device memory
    size_t max_size = 4000 * 4000 * sizeof(float);
    size_t size = n * d * sizeof(float);
    assert(size <= max_size);
    hipError_t err;

    if (!d_A) {
        err = hipMalloc(&d_A, max_size);
        //printf("CUDA malloc A: %s\n",hipGetErrorString(err));
        //printf("Copy A to device: %s\n",hipGetErrorString(err));
    }
    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    size = m * d * sizeof(float);
    assert(size <= max_size);
    if (!d_B) {
        err = hipMalloc(&d_B, max_size);
        //printf("CUDA malloc B: %s\n",hipGetErrorString(err));
        //printf("Copy B to device: %s\n",hipGetErrorString(err));
    }
    err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    size = n * m * sizeof(float);
    assert(size <= max_size);
    if (!d_C) {
        err = hipMalloc(&d_C, max_size);
        //printf("CUDA malloc C: %s\n",hipGetErrorString(err));
    }

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((m + dimBlock.x - 1) / dimBlock.x, (n + dimBlock.y - 1) / dimBlock.y);
    PairwiseL1Kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n, m, d);
    err = hipDeviceSynchronize();
    //printf("Run kernel: %s\n", hipGetErrorString(err));

    // Read C from device memory
    err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    //printf("Copy C off of device: %s\n",hipGetErrorString(err));

    // Free device memory
    // hipFree(d_A);
    // hipFree(d_B);
    // hipFree(d_C);
}

